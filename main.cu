#include "hip/hip_runtime.h"


#include <iostream>

__global__ void cuda_hello(){
    int i = threadIdx.x;
    int j = blockIdx.x;

    printf("Hello World from GPU!\n Thread: %d, Block: %d\n", i, j);
}

int main() {
    cuda_hello<<<3,10>>>(); 
    return 0;
}