#include "hip/hip_runtime.h"


#include <iostream>

__global__ void cuda_hello(){
    int i = threadIdx.x;
    int j = blockIdx.x;

    printf("Hello World from GPU!\n Thread: %d, Block: %d\n", i, j);
}

__global__ void matrix_addition(int *a, int *b, int *c, int N){
    int i = threadIdx.x;
    int j = blockIdx.x;

    // (i*N) + j is the index of the element in the 2D array
    c[i*N+j] = a[i*N+j] + b[i*N+j];
}

int main() {
    cuda_hello<<<2,2>>>(); 

    const int N = 3;
    int a[N][N], b[N][N], c[N][N];
    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, N*N*sizeof(int));
    hipMalloc((void**)&d_b, N*N*sizeof(int));
    hipMalloc((void**)&d_c, N*N*sizeof(int));

    // Initialize matrix a and b
    a[0][0] = 1; a[0][1] = 2; a[0][2] = 3;
    a[1][0] = 4; a[1][1] = 5; a[1][2] = 6;
    a[2][0] = 7; a[2][1] = 8; a[2][2] = 9;

    b[0][0] = 10; b[0][1] = 20; b[0][2] = 30;
    b[1][0] = 31; b[1][1] = 32; b[1][2] = 33;
    b[2][0] = 34; b[2][1] = 35; b[2][2] = 36;

    // Copy data from CPU to GPU
    hipMemcpy(d_a, a, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*N*sizeof(int), hipMemcpyHostToDevice);

    // Call kernel
    matrix_addition<<<3,3>>>(d_a, d_b, d_c, N);

    hipDeviceSynchronize();

    // Copy data from GPU to CPU
    hipMemcpy(c, d_c, N*N*sizeof(int), hipMemcpyDeviceToHost);

    // Print matrix a and b
    printf("Matrix A:\n");
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            printf("%d ", a[i][j]);
        }
        printf("\n");
    }

    printf("Matrix B:\n");
    for(int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            printf("%d ", b[i][j]);
        }
        printf("\n");
    }

    // Print result
    printf("Matrix C: (addition result) \n");
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 3; j++){
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}